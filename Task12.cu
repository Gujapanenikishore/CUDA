/*Task 12: CUDA Parallel Matrix Transpose (Team - 2 people)

Implement a matrix transpose operation using CUDA kernels.

Validate results rigorously against CPU-based matrix transpose.*/

#include <iostream>
#include <vector>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>

#define IDX2C(i, j, ld) (((j)*(ld))+(i))  // Column-major indexing if needed

// CUDA kernel for matrix transpose
__global__ void transposeKernel(float* out, const float* in, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // column
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    if (x < cols && y < rows)
        out[x * rows + y] = in[y * cols + x];  // Transpose
}

// CPU matrix transpose
void transposeCPU(std::vector<float>& out, const std::vector<float>& in, int rows, int cols) {
    for (int r = 0; r < rows; ++r)
        for (int c = 0; c < cols; ++c)
            out[c * rows + r] = in[r * cols + c];
}

// Validate equality
bool validate(const std::vector<float>& a, const std::vector<float>& b, float epsilon = 1e-5) {
    for (size_t i = 0; i < a.size(); ++i)
        if (fabs(a[i] - b[i]) > epsilon)
            return false;
    return true;
}

// Print matrix (row-major)
void printMatrix(const std::vector<float>& mat, int rows, int cols, const std::string& label) {
    std::cout << label << ":\n";
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c)
            std::cout << mat[r * cols + c] << "\t";
        std::cout << "\n";
    }
    std::cout << "\n";
}

int main() {
    const int rows = 4;
    const int cols = 4;
    const int size = rows * cols;

    std::vector<float> input = {
         1,  2,  3,  4,
         5,  6,  7,  8,
         9, 10, 11, 12,
        13, 14, 15, 16
    };
    std::vector<float> cpu_result(size), gpu_result(size);

    // CPU transpose
    transposeCPU(cpu_result, input, rows, cols);

    // Allocate CUDA memory
    float *d_in, *d_out;
    hipMalloc(&d_in, size * sizeof(float));
    hipMalloc(&d_out, size * sizeof(float));
    hipMemcpy(d_in, input.data(), size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(2, 2);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x,
                  (rows + blockSize.y - 1) / blockSize.y);

    // GPU transpose
    transposeKernel<<<gridSize, blockSize>>>(d_out, d_in, rows, cols);
    hipDeviceSynchronize();
    hipMemcpy(gpu_result.data(), d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    // Print results
    printMatrix(input, rows, cols, "Original Matrix");
    printMatrix(cpu_result, cols, rows, "CPU Transposed Matrix");
    printMatrix(gpu_result, cols, rows, "GPU Transposed Matrix");

    std::cout << "Validation: " << (validate(cpu_result, gpu_result) ? "PASS" : "FAIL") << "\n";

    return 0;
}

/*_______________________
OUTPUT

Original Matrix:
1	2	3	4
5	6	7	8
9	10	11	12
13	14	15	16

CPU Transposed Matrix:
1	5	9	13
2	6	10	14
3	7	11	15
4	8	12	16

GPU Transposed Matrix:
1	5	9	13
2	6	10	14
3	7	11	15
4	8	12	16

Validation: PASS   */